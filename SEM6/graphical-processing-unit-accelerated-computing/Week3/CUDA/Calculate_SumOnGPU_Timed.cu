#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void sum(int* a, int* b, int* c, const int N) {
        int i = threadIdx.x + blockDim.x * blockIdx.x;
        if (i < N) c[i] = a[i] + b[i];
        // else
        //	printf("i > N -> %3d in block: %d\n", i, blockIdx.x);
}

void sum_host(int* a, int* b, int* c, const int N) {
        clock_t clk;

        clk = clock();
        for (int i = 0; i < N; ++i) c[i] = a[i] + b[i];

        clk       = clock() - clk;
        double dt = (((double)clk) / CLOCKS_PER_SEC) * 1000;
        printf("Total time on CPU: %f msecs\n", dt);
}

int main() {
        int* h_a = 0;
        int* h_b = 0;
        int* h_c = 0;

        int* d_a = 0;
        int* d_b = 0;
        int* d_c = 0;

        const int N                  = 50000;  // 2048;
        int       numThreadsPerBlock = 128;
        int       numBlocksPerGrid   = ceilf(N / numThreadsPerBlock + 1);
        printf("Num threads per block: %3d\n", numThreadsPerBlock);
        printf("Num blocks per grid: %3d\n", numBlocksPerGrid);

        size_t size = N * sizeof(int);

        // allocate host memory
        h_a = (int*)malloc(size);
        h_b = (int*)malloc(size);
        h_c = (int*)malloc(size);

        // initialize a, b and c
        for (int i = 0; i < N; ++i) {
                h_a[i] = i + 1;
                h_b[i] = h_a[i] * 2;
                h_c[i] = 0;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // allocate device memory
        hipMalloc((void**)&d_a, size);
        hipMalloc((void**)&d_b, size);
        hipMalloc((void**)&d_c, size);

        // copy host data to device memory
        hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

        // calculate on host
        sum_host(h_a, h_b, h_c, N);
        printf("Sum (host): %d\n", h_c);

        // output result
        /*
        printf("Host calculation result: \n");
        for(int i=0;i<N;++i) {
            printf("%3d + %3d = %3d\n", h_a[i], h_b[i], h_c[i]);
            //clear host result to ensure that the result of device is actually from
        the kernel h_c[i] = 0;
        }
        */

        hipEventRecord(start);
        // calculate on device
        sum<<<numBlocksPerGrid, numThreadsPerBlock>>>(d_a, d_b, d_c, N);
        hipDeviceSynchronize();
        hipEventRecord(stop);

        // copy result from device to host
        hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

        hipEventSynchronize(stop);

        printf("Sum (device): %d\n", h_c);

        // output result
        /*
        printf("--------------------------------------\n");
        printf("Device calculation result: \n");
        for(int i=0;i<N;++i) {
            printf("%3d + %3d = %3d\n", h_a[i], h_b[i], h_c[i]);
        }
        printf("--------------------------------------\n");
        */

        float dt = 0;
        hipEventElapsedTime(&dt, start, stop);
        printf("Total time on GPU: %f msecs\n", dt);

        // delete data allocated on device
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        // delete host memory
        free(h_a);
        free(h_b);
        free(h_c);

        hipDeviceReset();
        return 0;
}