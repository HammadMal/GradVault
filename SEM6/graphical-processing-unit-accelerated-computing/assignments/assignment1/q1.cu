#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 10000000

/**
 * @brief Check for CUDA errors
 *
 * @param err
 * @param msg
 * @return cudaError_t
 */
inline hipError_t checkCudaErr(hipError_t err, const char *msg)
{
        if (err != hipSuccess)
        {
                fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
        }
        return err;
}

/**
 * @brief Initialize the random number generator
 *
 * @param state
 * @param seed
 */
__global__ void init_rand(hiprandState *state, unsigned long seed)
{
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(seed, id, 0, &state[id]);
}

/**
 * @brief Generate random numbers on GPU
 *
 * @param state
 * @param rand
 */
__global__ void generate_rand(hiprandState *state, float *rand)
{
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        rand[id] = hiprand_uniform(&state[id]);
}

int main()
{
        float *rande, *d_rand, cpu_time_used, gpu_time_used, memcpy_time_used;
        int size = N * sizeof(float),             // CPU random number array size
            state_size = N * sizeof(hiprandState); // GPU random number array size
        hiprandState *state, *d_state;             // GPU random number generator state array
        clock_t start, end;                       // CPU timers
        hipEvent_t start_gpu, stop_gpu;          // GPU timers

        rande = (float *)malloc(size);
        state = (hiprandState *)malloc(state_size);

        checkCudaErr(hipMalloc((void **)&d_rand, size), "hipMalloc");
        checkCudaErr(hipMalloc((void **)&d_state, state_size), "hipMalloc");

        start = clock();
        for (int i = 0; i < N; i++)
                rande[i] = (float)rand() / (float)RAND_MAX;
        end = clock();
        cpu_time_used = ((float)(end - start)) / CLOCKS_PER_SEC;
        printf("CPU time used:    %10f seconds to generate random numbers on CPU \n", cpu_time_used);

        checkCudaErr(hipEventCreate(&start_gpu), "hipEventCreate");
        checkCudaErr(hipEventCreate(&stop_gpu), "hipEventCreate");
        checkCudaErr(hipEventRecord(start_gpu, 0), "hipEventRecord");

        init_rand<<<N / 256, 256>>>(d_state, time(NULL));
        generate_rand<<<N / 256, 256>>>(d_state, d_rand);

        checkCudaErr(hipEventRecord(stop_gpu, 0), "hipEventRecord");
        checkCudaErr(hipEventSynchronize(stop_gpu), "hipEventSynchronize");

        checkCudaErr(hipEventElapsedTime(&gpu_time_used, start_gpu, stop_gpu), "hipEventElapsedTime");

        printf("GPU time used:    %10f seconds to generate random numbers on GPU \n", gpu_time_used / 1000.0f);

        start = clock();

        checkCudaErr(hipMemcpy(rande, d_rand, size, hipMemcpyDeviceToHost), "hipMemcpy");
        end = clock();
        memcpy_time_used = ((float)(end - start)) / CLOCKS_PER_SEC;
        printf("MEMCPY time used: %10f seconds to copy random numbers from GPU to "
               "CPU\n",
               memcpy_time_used / 1000.0f);

        checkCudaErr(hipFree(d_rand), "hipFree");
        checkCudaErr(hipFree(d_state), "hipFree");

        free(rande);
        free(state);

        return 0;
}