#include <hip/hip_runtime.h>
#include <stdio.h>

inline hipError_t checkCudaErr(hipError_t err, const char* msg) {
        if (err != hipSuccess) {
                fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
        }
        return err;
}

__global__ void InitData(int* data) { data[threadIdx.x] += threadIdx.x; }

int main() {
        const int N = 100;

        int* h_data = (int*)malloc(N * sizeof(int));
        int* d_data;

        // allocate memory on device
        hipMalloc((void**)&d_data, N * sizeof(int));

        // call kernel
        InitData<<<1, N>>>(d_data);

        checkCudaErr(hipDeviceSynchronize(), "Syncronization");
        checkCudaErr(hipGetLastError(), "GPU Error");

        // copy device data to host memory
        checkCudaErr(hipMemcpy(h_data, d_data, N * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy D->H");

        printf("Data (on device): \n");
        for (int i = 0; i < N; ++i) printf(" data[%d] -> %d\n", i, h_data[i]);

        // release GPU memory
        hipFree(d_data);
        free(h_data);

        return 0;
}