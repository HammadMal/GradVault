
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

inline hipError_t checkCudaErr(hipError_t err, const char *msg) {
        if (err != hipSuccess) {
                fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
        }
        return err;
}

// kernel function definition
__global__ void add(int *a, int *b, int *c) { *c = *a + *b; }

int main() {
        int h_a = 10;
        int h_b = 20;
        int h_c = 10 + 20;

        int *d_a, *d_b, *d_c;

        // allocate memory on device
        hipMalloc((void **)&d_a, sizeof(int));
        hipMalloc((void **)&d_b, sizeof(int));
        hipMalloc((void **)&d_c, sizeof(int));

        // copy host data to device memory
        hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

        // call kernel
        add<<<1, 1>>>(d_a, d_b, d_c);

        checkCudaErr(hipDeviceSynchronize(), "Syncronization");
        checkCudaErr(hipGetLastError(), "GPU");
        printf("Answer (on host): %d\n", h_c);

        // copy device data to host memory
        hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
        printf("Answer (on device): %d", h_c);

        // release GPU memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        return 0;
}